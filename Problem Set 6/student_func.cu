#include "hip/hip_runtime.h"
//Udacity HW 6
//Poisson Blending

/* Background
   ==========

   The goal for this assignment is to take one image (the source) and
   paste it into another image (the destination) attempting to match the
   two images so that the pasting is non-obvious. This is
   known as a "seamless clone".

   The basic ideas are as follows:

   1) Figure out the interior and border of the source image
   2) Use the values of the border pixels in the destination image 
      as boundary conditions for solving a Poisson equation that tells
      us how to blend the images.
   
      No pixels from the destination except pixels on the border
      are used to compute the match.

   Solving the Poisson Equation
   ============================

   There are multiple ways to solve this equation - we choose an iterative
   method - specifically the Jacobi method. Iterative methods start with
   a guess of the solution and then iterate to try and improve the guess
   until it stops changing.  If the problem was well-suited for the method
   then it will stop and where it stops will be the solution.

   The Jacobi method is the simplest iterative method and converges slowly - 
   that is we need a lot of iterations to get to the answer, but it is the
   easiest method to write.

   Jacobi Iterations
   =================

   Our initial guess is going to be the source image itself.  This is a pretty
   good guess for what the blended image will look like and it means that
   we won't have to do as many iterations compared to if we had started far
   from the final solution.

   ImageGuess_prev (Floating point)
   ImageGuess_next (Floating point)

   DestinationImg
   SourceImg

   Follow these steps to implement one iteration:

   1) For every pixel p in the interior, compute two sums over the four neighboring pixels:
      Sum1: If the neighbor is in the interior then += ImageGuess_prev[neighbor]
             else if the neighbor in on the border then += DestinationImg[neighbor]

      Sum2: += SourceImg[p] - SourceImg[neighbor]   (for all four neighbors)

   2) Calculate the new pixel value:
      float newVal= (Sum1 + Sum2) / 4.f  <------ Notice that the result is FLOATING POINT
      ImageGuess_next[p] = min(255, max(0, newVal)); //clamp to [0, 255]


    In this assignment we will do 800 iterations.
   */

#include "utils.h"
#include ""
#include <stdio.h>
#include <vector>

const int block_x = 16;
const int block_y = 16;
const int filterWidth = 3;

__device__ void clamp(int & pos, int maxpos) {

	pos = pos > 0 ? pos : 0;
	pos = pos < (maxpos - 1) ? pos : (maxpos - 1);

}



__device__ bool mask(uchar4 val) {

	return (val.x != 255 || val.y != 255 || val.z != 255);

}


__global__ void bord_int(const uchar4* const d_sourceImg, const size_t numRowsSource, const size_t numColsSource, unsigned char* border, 														unsigned char* interior, int* 															xcoords, int* ycoords) {
	
	 __shared__ uchar4 sh_arr[(block_x + filterWidth - 1)*(block_y + filterWidth - 1)];

	//Load data in shared mem

	const int2 make_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
						blockIdx.y * blockDim.y + threadIdx.y);


	int make_1D_pos = make_2D_pos.y * numColsSource + make_2D_pos.x;
	int halfWidth = filterWidth/2; 

	int load_x_new = make_2D_pos.x - halfWidth;
	int load_y_new = make_2D_pos.y - halfWidth;

	clamp(load_x_new, numColsSource);
	clamp(load_y_new, numRowsSource);

	int load_x_original = load_x_new;
	int load_y_original = load_y_new;

	sh_arr[threadIdx.y*(blockDim.x + filterWidth - 1) + threadIdx.x] = d_sourceImg[load_y_new*numColsSource + load_x_new];

	if (threadIdx.y >= (blockDim.y - filterWidth + 1)) {

		load_y_new = make_2D_pos.y + halfWidth;
		clamp(load_y_new, numRowsSource);
		sh_arr[(threadIdx.y + filterWidth - 1)*(blockDim.x + filterWidth - 1) + threadIdx.x] = d_sourceImg[load_y_new*numColsSource + 																	load_x_original];

	}	

	if (threadIdx.x >= (blockDim.x - filterWidth + 1)) {

		load_x_new = make_2D_pos.x + halfWidth;
		clamp(load_x_new, numColsSource);
		sh_arr[(threadIdx.y)*(blockDim.x + filterWidth - 1) + threadIdx.x + filterWidth - 1] = d_sourceImg[load_y_original*numColsSource + load_x_new];

	}
	if (threadIdx.x < (filterWidth - 1) && threadIdx.y < (filterWidth - 1)) {

		load_x_new = make_2D_pos.x - halfWidth + blockDim.x;
		load_y_new = make_2D_pos.y - halfWidth + blockDim.y;
		clamp(load_x_new, numColsSource);
		clamp(load_y_new, numRowsSource);
		sh_arr[(threadIdx.y + blockDim.y)*(blockDim.x + filterWidth - 1) + threadIdx.x + blockDim.x] = d_sourceImg[load_y_new*numColsSource + load_x_new];

	}

	//End load data
	__syncthreads();

	if (make_2D_pos.x >= numColsSource ||
		make_2D_pos.y >= numRowsSource) return;
	
	int target_pos_x = threadIdx.x + halfWidth;
	int target_pos_y = threadIdx.y + halfWidth;
	if (!mask(sh_arr[target_pos_y*(blockDim.x + filterWidth - 1) + target_pos_x]))return; //neither interior or border



	if (!mask(sh_arr[target_pos_y*(blockDim.x + filterWidth - 1) + target_pos_x-1]) || !mask(sh_arr[target_pos_y*(blockDim.x + filterWidth - 1) + target_pos_x+1]) ||
			!mask(sh_arr[(target_pos_y-1)*(blockDim.x + filterWidth - 1) + target_pos_x]) || !mask(sh_arr[(target_pos_y+1)*(blockDim.x + filterWidth - 1) + 																			target_pos_x])) {

		border[make_1D_pos] = 1;
		xcoords[make_1D_pos] = make_2D_pos.x;
		ycoords[make_1D_pos] = make_2D_pos.y;
		return;

	}
	
	interior[make_1D_pos] = 1;

}


__device__ float _min(float a, float b) {

	if (a < 0)a = 999999;
	if (b < 0)b = 999999;
	return (a < b) ? a : b;

}

__device__ float _max(float a, float b) {

	return a > b ? a : b;

}

__global__ void minmax_reduce(int* d_out, const int* d_in, int input_size, bool isMin) {

	extern __shared__ float sdata[];

	int t_id = threadIdx.x;
	int global_id = t_id + blockDim.x*blockIdx.x;

	if (global_id >= input_size) { 
		sdata[t_id] = d_in[0]; 
	} 

	else sdata[t_id] = d_in[global_id];

	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1) {

		if (t_id < s) sdata[t_id] = isMin ? _min(sdata[t_id], sdata[t_id + s]) : _max(sdata[t_id], sdata[t_id + s]);

		__syncthreads();

	}

	if (t_id == 0) {

		d_out[blockIdx.x] = sdata[0];

	}
}

int reduce(const int* const d_in, int input_size, bool isMin) {
	int threads = block_x*block_y;
	int* d_current_in = NULL;
	int size = input_size;
	int blocks = ceil(1.0f*size / threads);
	while (true) {

		//allocate memory for intermediate results
		int* d_out;
		checkCudaErrors(hipMalloc(&d_out, blocks * sizeof(int)));

		//call reduce kernel
		if (d_current_in == NULL) minmax_reduce << <blocks, threads, threads * sizeof(int) >> > (d_out, d_in, size, isMin);
		else minmax_reduce << <blocks, threads, threads * sizeof(int) >> > (d_out, d_current_in, size, isMin);;
		hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

		//free last intermediate result
		if (d_current_in != NULL) checkCudaErrors(hipFree(d_current_in));

		if (blocks == 1) {
			
			int h_out;
			checkCudaErrors(hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost));
			return h_out;
		}
		size = blocks;
		blocks = ceil(1.0f*size / threads);
		if (blocks == 0)blocks++;
		d_current_in = d_out;
	}

}

//This function takes an image in uchar4 and splits it into three different colored images
__global__
void separate_channels(const uchar4* const inputImageRGBA,
	int numRows,
	int numCols,
	float* const red_channel,
	float* const green_channel,
	float* const blue_channel)
{

	const int2 make_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	if (make_2D_pos.x >= numCols ||
		make_2D_pos.y >= numRows)
	{
		return;
	}
	int make_1D_pos = make_2D_pos.y * numCols + make_2D_pos.x;
	red_channel[make_1D_pos] = (float)inputImageRGBA[make_1D_pos].x;
	green_channel[make_1D_pos] = (float)inputImageRGBA[make_1D_pos].y;
	blue_channel[make_1D_pos] = (float)inputImageRGBA[make_1D_pos].z;
}

__global__
void jacobi(float* const d_original_in,float* const d_in, float* const d_source_in, 					   unsigned char *border, unsigned char *interior,  					   float* d_out, int minx,int miny, 				           int numRowsSource,int numColsSource) {

	__shared__ float sh_arr_source[(block_x + filterWidth - 1)*(block_y + filterWidth - 1)];
	__shared__ float sh_arr_target[(block_x + filterWidth - 1)*(block_y + filterWidth - 1)];
	__shared__ float sh_arr_target_original[(block_x + filterWidth - 1)*(block_y + filterWidth - 1)];
	__shared__ unsigned char sh_interior[(block_x + filterWidth - 1)*(block_y + filterWidth - 1)];
	__shared__ unsigned char sh_border[(block_x + filterWidth - 1)*(block_y + filterWidth - 1)];
	//lopad data in shared mem

	const int2 make_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x+minx,
		blockIdx.y * blockDim.y + threadIdx.y+miny);


	int make_1D_pos = make_2D_pos.y * numColsSource + make_2D_pos.x;
	int halfWidth = filterWidth / 2;

	int load_x_new = make_2D_pos.x - halfWidth;
	int load_y_new = make_2D_pos.y - halfWidth;

	clamp(load_x_new, numColsSource);
	clamp(load_y_new, numRowsSource);

	int load_x_original = load_x_new;
	int load_y_original = load_y_new;

	sh_arr_source[threadIdx.y*(blockDim.x + filterWidth - 1) + threadIdx.x] = d_source_in[load_y_new*numColsSource + load_x_new];
	sh_arr_target[threadIdx.y*(blockDim.x + filterWidth - 1) + threadIdx.x] = d_in[load_y_new*numColsSource + load_x_new];
	sh_arr_target_original[threadIdx.y*(blockDim.x + filterWidth - 1) + threadIdx.x] = d_original_in[load_y_new*numColsSource + load_x_new];
	sh_interior[threadIdx.y*(blockDim.x + filterWidth - 1) + threadIdx.x] = interior[load_y_new*numColsSource + load_x_new];
	sh_border[threadIdx.y*(blockDim.x + filterWidth - 1) + threadIdx.x] = border[load_y_new*numColsSource + load_x_new];

	if (threadIdx.y >= (blockDim.y - filterWidth + 1)) {
		load_y_new = make_2D_pos.y + halfWidth;
		clamp(load_y_new, numRowsSource);
		sh_arr_source[(threadIdx.y + filterWidth - 1)*(blockDim.x + filterWidth - 1) + threadIdx.x] = d_source_in[load_y_new*numColsSource + load_x_original];
		sh_arr_target[(threadIdx.y + filterWidth - 1)*(blockDim.x + filterWidth - 1) + threadIdx.x] = d_in[load_y_new*numColsSource + load_x_original];
		sh_arr_target_original[(threadIdx.y + filterWidth - 1)*(blockDim.x + filterWidth - 1) + threadIdx.x] = d_original_in[load_y_new*numColsSource + load_x_original];
		sh_interior[(threadIdx.y + filterWidth - 1)*(blockDim.x + filterWidth - 1) + threadIdx.x] = interior[load_y_new*numColsSource + load_x_original];
		sh_border[(threadIdx.y + filterWidth - 1)*(blockDim.x + filterWidth - 1) + threadIdx.x] = border[load_y_new*numColsSource + load_x_original];
	}
	
	if (threadIdx.x >= (blockDim.x - filterWidth + 1)) {

		load_x_new = make_2D_pos.x + halfWidth;
		clamp(load_x_new, numColsSource);

		sh_arr_source[(threadIdx.y)*(blockDim.x + filterWidth - 1) + threadIdx.x + filterWidth - 1] = d_source_in[load_y_original*numColsSource + load_x_new];
		sh_arr_target[(threadIdx.y)*(blockDim.x + filterWidth - 1) + threadIdx.x + filterWidth - 1] = d_in[load_y_original*numColsSource + load_x_new];
		sh_arr_target_original[(threadIdx.y)*(blockDim.x + filterWidth - 1) + threadIdx.x + filterWidth - 1] = d_original_in[load_y_original*numColsSource + 																					load_x_new];
		sh_interior[(threadIdx.y)*(blockDim.x + filterWidth - 1) + threadIdx.x + filterWidth - 1] = interior[load_y_original*numColsSource + load_x_new];
		sh_border[(threadIdx.y)*(blockDim.x + filterWidth - 1) + threadIdx.x + filterWidth - 1] = border[load_y_original*numColsSource + load_x_new];
	}

	if (threadIdx.x < (filterWidth - 1) && threadIdx.y < (filterWidth - 1)) {

		load_x_new = make_2D_pos.x - halfWidth + blockDim.x;
		load_y_new = make_2D_pos.y - halfWidth + blockDim.y;
		clamp(load_x_new, numColsSource);
		clamp(load_y_new, numRowsSource);

		sh_arr_source[(threadIdx.y + blockDim.y)*(blockDim.x + filterWidth - 1) + threadIdx.x + blockDim.x] = d_source_in[load_y_new*numColsSource + load_x_new];
		sh_arr_target[(threadIdx.y + blockDim.y)*(blockDim.x + filterWidth - 1) + threadIdx.x + blockDim.x] = d_in[load_y_new*numColsSource + load_x_new];
		sh_arr_target_original[(threadIdx.y + blockDim.y)*(blockDim.x + filterWidth - 1) + threadIdx.x + blockDim.x] = d_original_in[load_y_new*numColsSource + 																				load_x_new];
		sh_interior[(threadIdx.y + blockDim.y)*(blockDim.x + filterWidth - 1) + threadIdx.x + blockDim.x] = interior[load_y_new*numColsSource + load_x_new];
		sh_border[(threadIdx.y + blockDim.y)*(blockDim.x + filterWidth - 1) + threadIdx.x + blockDim.x] = border[load_y_new*numColsSource + load_x_new];
	}

	//end load data
	__syncthreads();

	
	if (make_2D_pos.x >= numColsSource ||
		make_2D_pos.y >= numRowsSource) return;
	
	if (interior[make_1D_pos] != 1)return; //filter out boundary points

	int target_pos_x = threadIdx.x + halfWidth;
	int target_pos_y = threadIdx.y + halfWidth;
	float valsource = sh_arr_source[target_pos_y*(blockDim.x + filterWidth - 1) + target_pos_x];

	float sum1=0.0f;
	float sum2 = 4 * valsource;

	int curpos[] = { target_pos_y*(blockDim.x + filterWidth - 1) + target_pos_x - 1,target_pos_y*(blockDim.x + filterWidth - 1) + target_pos_x + 1,
						(target_pos_y - 1)*(blockDim.x + filterWidth - 1) + target_pos_x,(target_pos_y + 1)*(blockDim.x + filterWidth - 1) + target_pos_x };

	for (int i = 0; i < 4; i++) {
		if (sh_interior[curpos[i]]) {
			sum1 += sh_arr_target[curpos[i]];
			
		}
		else if (sh_border[curpos[i]]) {
			sum1 += sh_arr_target_original[curpos[i]];
			
		}
		sum2 -= sh_arr_source[curpos[i]];
	}
	
	float newVal = (sum1 + sum2) / 4.0f;
	newVal = newVal < 0 ? 0 : newVal;
	newVal = newVal > 255 ? 255 : newVal;

	d_out[make_1D_pos] = newVal;
}

__global__
void recomb_channel(const uchar4* const d_destImg,const float* const red_channel,
	const float* const green_channel,
	const float* const blue_channel,
	uchar4* const outputImageRGBA,
	int numRows,
	int numCols,unsigned char* interior)
{
	const int2 make_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	const int make_1D_pos = make_2D_pos.y * numCols + make_2D_pos.x;

	//doesn't access memory outside of image
	if (make_2D_pos.x >= numCols || make_2D_pos.y >= numRows)
		return;

	if (interior[make_1D_pos] != 1) {
		outputImageRGBA[make_1D_pos] = d_destImg[make_1D_pos];
		return;
	}

	unsigned char red = (unsigned char)red_channel[make_1D_pos];
	unsigned char green = (unsigned char)green_channel[make_1D_pos];
	unsigned char blue = (unsigned char)blue_channel[make_1D_pos];

	//alpha is 255
	uchar4 outputPixel = make_uchar4(red, green, blue, 255);
	
	outputImageRGBA[make_1D_pos] = outputPixel;
}


void your_blend(const uchar4* const h_sourceImg,  //IN
                const size_t numRowsSource, const size_t numColsSource,
                const uchar4* const h_destImg, //IN
                uchar4* const h_blendedImg) //OUT
{


	const dim3 blockSize(block_x, block_y);
	const dim3 gridSize(ceil(1.0f*numColsSource / blockSize.x), ceil(1.0f*numRowsSource / blockSize.y));

	
	

	uchar4* d_sourceImg;
	unsigned char *border,*interior;
	int *xcoords, *ycoords; //for bounding box computation

	checkCudaErrors(hipMalloc(&d_sourceImg, numRowsSource*numColsSource * sizeof(uchar4)));
	checkCudaErrors(hipMalloc(&border, numRowsSource*numColsSource * sizeof(unsigned char)));
	checkCudaErrors(hipMalloc(&interior, numRowsSource*numColsSource * sizeof(unsigned char)));
	checkCudaErrors(hipMalloc(&xcoords, numRowsSource*numColsSource * sizeof(int)));
	checkCudaErrors(hipMalloc(&ycoords, numRowsSource*numColsSource * sizeof(int)));
	checkCudaErrors(hipMemset(border, 0, numRowsSource*numColsSource * sizeof(unsigned char)));
	checkCudaErrors(hipMemset(interior, 0, numRowsSource*numColsSource * sizeof(unsigned char)));
	checkCudaErrors(hipMemset(xcoords, -1, numRowsSource*numColsSource * sizeof(int)));
	checkCudaErrors(hipMemset(ycoords, -1, numRowsSource*numColsSource * sizeof(int)));
	checkCudaErrors(hipMemcpy(d_sourceImg, h_sourceImg, numRowsSource*numColsSource * sizeof(uchar4), hipMemcpyHostToDevice));

	bord_int << <gridSize, blockSize >> > (d_sourceImg, numRowsSource, numColsSource, border, interior, xcoords, ycoords);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	
	int minx = reduce(xcoords, numRowsSource*numColsSource, true);
	int maxx = reduce(xcoords, numRowsSource*numColsSource, false);
	int miny = reduce(ycoords, numRowsSource*numColsSource, true);
	int maxy = reduce(ycoords, numRowsSource*numColsSource, false);

	int size_x = maxx - minx+1;
	int size_y = maxy - miny + 1;

	checkCudaErrors(hipFree(xcoords));
	checkCudaErrors(hipFree(ycoords));

	
	

	uchar4* d_destImg;

	checkCudaErrors(hipMalloc(&d_destImg, numRowsSource*numColsSource * sizeof(uchar4)));
	checkCudaErrors(hipMemcpy(d_destImg, h_destImg, numRowsSource*numColsSource * sizeof(uchar4), hipMemcpyHostToDevice));

	float *d_buffer_red_1, *d_buffer_red_2;
	float *d_buffer_green_1, *d_buffer_green_2;
	float *d_buffer_blue_1, *d_buffer_blue_2;
	float *d_red, *d_green, *d_blue;

	checkCudaErrors(hipMalloc(&d_buffer_red_1, numRowsSource*numColsSource * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_buffer_red_2, numRowsSource*numColsSource * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_buffer_green_1, numRowsSource*numColsSource * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_buffer_green_2, numRowsSource*numColsSource * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_buffer_blue_1, numRowsSource*numColsSource * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_buffer_blue_2, numRowsSource*numColsSource * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_red, numRowsSource*numColsSource * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_green, numRowsSource*numColsSource * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_blue, numRowsSource*numColsSource * sizeof(float)));
	
	separate_channels << <gridSize, blockSize >> > (d_destImg, numRowsSource, numColsSource, d_red, d_green, d_blue);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	
	
	float *d_red_source, *d_green_source, *d_blue_source;

	checkCudaErrors(hipMalloc(&d_red_source, numRowsSource*numColsSource * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_green_source, numRowsSource*numColsSource * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_blue_source, numRowsSource*numColsSource * sizeof(float)));
	separate_channels << <gridSize, blockSize >> > (d_sourceImg, numRowsSource, numColsSource, d_red_source, d_green_source, d_blue_source);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	hipStream_t s1, s2, s3;
	hipStreamCreate(&s1); hipStreamCreate(&s2); hipStreamCreate(&s3);

	checkCudaErrors(hipMemcpyAsync(d_buffer_red_1, d_red_source, numRowsSource*numColsSource * sizeof(float), hipMemcpyDeviceToDevice,s1));
	checkCudaErrors(hipMemcpyAsync(d_buffer_green_1, d_green_source, numRowsSource*numColsSource * sizeof(float), hipMemcpyDeviceToDevice,s2));
	checkCudaErrors(hipMemcpyAsync(d_buffer_blue_1, d_blue_source, numRowsSource*numColsSource * sizeof(float), hipMemcpyDeviceToDevice,s3));


	
	

	const dim3 gridSizeNew(ceil(1.0f*size_x / blockSize.x), ceil(1.0f*size_y / blockSize.y));
	
	for (size_t i = 0; i < 800; i++) {

		if (i % 2 == 0) {

			//source is buffer 1
			jacobi << <blockSize, gridSizeNew,0,s1 >> > (d_red,d_buffer_red_1,d_red_source,border,interior,d_buffer_red_2,minx,miny,numRowsSource,numColsSource);
			jacobi << <blockSize, gridSizeNew,0,s2 >> > (d_green, d_buffer_green_1, d_green_source, border, interior, d_buffer_green_2, minx, miny, numRowsSource, 																					numColsSource);
			jacobi << <blockSize, gridSizeNew,0,s3 >> > (d_blue, d_buffer_blue_1, d_blue_source, border, interior, d_buffer_blue_2, minx, miny, numRowsSource, 																					numColsSource);
		}

		else {
			//source is buffer 2
			jacobi << <blockSize, gridSizeNew,0,s1 >> > (d_red, d_buffer_red_2, d_red_source, border, interior, d_buffer_red_1, minx, miny, numRowsSource, 																					numColsSource);
			jacobi << <blockSize, gridSizeNew,0,s2 >> > (d_green, d_buffer_green_2, d_green_source, border, interior, d_buffer_green_1, minx, miny, numRowsSource, 																					numColsSource);
			jacobi << <blockSize, gridSizeNew,0,s3 >> > (d_blue, d_buffer_blue_2, d_blue_source, border, interior, d_buffer_blue_1, minx, miny, numRowsSource, 																				numColsSource);		
		}
		
	}

	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	hipStreamDestroy(s1); hipStreamDestroy(s2); hipStreamDestroy(s3);

	uchar4* d_blendedImg;

	checkCudaErrors(hipMalloc(&d_blendedImg, numRowsSource*numColsSource * sizeof(uchar4)));
	recomb_channel << <blockSize, gridSize >> > (d_destImg,d_buffer_red_1, d_buffer_green_1, d_buffer_blue_1, d_blendedImg, numRowsSource, numColsSource,interior);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemcpy(h_blendedImg, d_blendedImg, numRowsSource*numColsSource * sizeof(uchar4), hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(d_sourceImg));
	checkCudaErrors(hipFree(d_destImg));
	checkCudaErrors(hipFree(border));
	checkCudaErrors(hipFree(interior));
	checkCudaErrors(hipFree(d_red));
	checkCudaErrors(hipFree(d_green));
	checkCudaErrors(hipFree(d_blue));
	checkCudaErrors(hipFree(d_red_source));
	checkCudaErrors(hipFree(d_green_source));
	checkCudaErrors(hipFree(d_blue_source));
	checkCudaErrors(hipFree(d_buffer_red_1));
	checkCudaErrors(hipFree(d_buffer_red_2));
	checkCudaErrors(hipFree(d_buffer_green_1));
	checkCudaErrors(hipFree(d_buffer_green_2));
	checkCudaErrors(hipFree(d_buffer_blue_1));
	checkCudaErrors(hipFree(d_buffer_blue_2));
}

	
	


				

     


	
